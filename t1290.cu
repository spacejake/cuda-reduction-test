#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <math.h>

__inline__ __device__
float warpReduceSum(float val) {
    for (int offset = warpSize / 2; offset > 0; offset /= 2)
        val += __shfl_down(val, offset);
    return val;
}

__inline__ __device__
float blockReduceSum(float val) {

    static __shared__ int shared[32]; // Shared mem for 32 partial sums
    int lane = threadIdx.x % warpSize;
    int wid = threadIdx.x / warpSize;

    val = warpReduceSum(val);     // Each warp performs partial reduction

    if (lane == 0) shared[wid] = val; // Write reduced value to shared memory

    __syncthreads();              // Wait for all partial reductions

    //read from shared memory only if that warp existed
    val = (threadIdx.x < blockDim.x / warpSize) ? shared[lane] : 0;

    if (wid == 0) val = warpReduceSum(val); //Final reduce within first warp

    return val;
}

__global__ void deviceReduceKernel(float *in, float *out, size_t N) {
    float sum = 0;
    //reduce multiple elements per thread
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x) {
        sum += in[i];
    }
    sum = blockReduceSum(sum);
    if (threadIdx.x == 0)
        out[blockIdx.x] = sum;
}

__global__ void deviceReduceWarpAtomicKernel(float *in, float *out, int N) {
    float sum = float(0);
    for (int i = blockIdx.x * blockDim.x + threadIdx.x;
         i < N;
         i += blockDim.x * gridDim.x) {

        sum += in[i] + in[i + 1] + in[i + 2];
    }
    sum = warpReduceSum(sum);
    if ((threadIdx.x & (warpSize - 1)) == 0)
        atomicAdd(out, sum);
}

__global__ void deviceReduceBlockAtomicKernel(float *in, float *out, float *out_j, int N, int rank) {
    float sum = float(0);
    for (int i = blockIdx.x * blockDim.x + threadIdx.x;
         i < N;
         i += blockDim.x * gridDim.x) {
        float sum_i = in[i] + in[i + 1] + in[i + 2];
        sum += sum_i;

        for (int j = 0; j < rank; j++) {
            float jacobi = -2.0 * sum_i;

            jacobi = blockReduceSum(jacobi);

            if (threadIdx.x == 0)
                atomicAdd(&out_j[j], jacobi);
        }
    }
    sum = blockReduceSum(sum);
    if (threadIdx.x == 0)
        atomicAdd(out, sum);
}

int main() {
    int n = 5000;
    float b;
    float *a, *a_host;
    int rank = 10;
    int size = n * 3;
    float j[rank];
    a_host = new float[size];

    int blocks = (n / 512) + 1;

    float *d_b;
    hipMalloc(&d_b, sizeof(float));

    float *d_j;
    hipMalloc(&d_j, rank * sizeof(float));

    hipMalloc((void **) &a, size * sizeof(float));
    for (int i = 0; i < n; i++) {
        a_host[i] = 1;
        a_host[i + 1] = 1;
        a_host[i + 2] = 1;
    }

    hipMemcpy(a, a_host, size * sizeof(float), hipMemcpyHostToDevice);

    deviceReduceBlockAtomicKernel << < blocks, 512 >> > (a, d_b, d_j, n, rank);
    //deviceReduceKernel<<<1, 1024>>>(d_intermediate, a, blocks);
    hipMemcpy(&b, d_b, sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&j, d_j, rank*sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_b);
    hipFree(d_j);
    std::cout << "Result: " << b << std::endl;
    std::cout << "Jacobi:";
    for( int i = 1; i < rank; i++) {
        std::cout << " " << j[i];
    }
    std::cout << std::endl;

    return 0;
}
